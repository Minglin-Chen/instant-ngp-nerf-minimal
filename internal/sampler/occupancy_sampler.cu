#include "hip/hip_runtime.h"
/** @author	Minglin Chen
 *  @date 	2023/3/23
 *  @Email 	chenmlin8@mail2.sysu.edu.cn
 */

#include "internal/utils/common.h"
#include "internal/sampler/bounding_box.h"
#include "internal/utils/random_val.cuh"
#include "internal/testbed.h"
#include "internal/utils/envmap.h"

using namespace Eigen;
using namespace tcnn;


inline constexpr __device__ float SQRT3() { return 1.73205080757f; }

__global__ void mark_untrained_density_grid(
	const uint32_t n_elements, 
	const uint32_t n_grid_size,
	const uint32_t n_images,
	const Vector2i resolution,
	const Vector2f focal_length,
	const Matrix<float, 3, 4>* training_xforms,
	float* __restrict__ grid_out
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	uint32_t level = i / (n_grid_size*n_grid_size*n_grid_size);
	uint32_t pos_idx = i % (n_grid_size*n_grid_size*n_grid_size);

	uint32_t x = tcnn::morton3D_invert(pos_idx>>0);
	uint32_t y = tcnn::morton3D_invert(pos_idx>>1);
	uint32_t z = tcnn::morton3D_invert(pos_idx>>2);

	Vector3f pos = (
		(Vector3f{(float)x+0.5f, (float)y+0.5f, (float)z+0.5f}) / n_grid_size - Vector3f::Constant(0.5f)) * scalbnf(1.0f, level) + Vector3f::Constant(0.5f);

	float half_resx = resolution.x() * 0.5f;
	float half_resy = resolution.y() * 0.5f;
	float voxel_radius = 0.5f*SQRT3()*scalbnf(1.0f, level) / n_grid_size;

	int count = 0;
	for (uint32_t j=0; j<n_images; ++j) {
		Matrix<float, 3, 4> xform = training_xforms[j];
		Vector3f ploc = pos - xform.col(3);
		float x = ploc.dot(xform.col(0));
		float y = ploc.dot(xform.col(1));
		float z = ploc.dot(xform.col(2));
		if (z > 0.f) {
			auto focal = focal_length;
			// TODO - add a box / plane intersection to stop thomas from murdering me
			if (fabsf(x)-voxel_radius < z/focal.x()*half_resx && fabsf(y)-voxel_radius < z/focal.y()*half_resy) {
				count++;
				if (count > 0) break;
			}
		}
	}
	grid_out[i] = (count > 0) ? 0.f : -1.f;
}

__global__ void generate_grid_samples_nerf_nonuniform(
	const uint32_t n_elements, 
	default_rng_t rng, 
	const uint32_t step, 
	const BoundingBox aabb, 
	const float* __restrict__ grid_in, 
	Position* __restrict__ out, 
	uint32_t* __restrict__ indices, 
	const uint32_t n_grid_size,
	const uint32_t n_grid_elements,
	const uint32_t n_cascades, 
	const float min_cone_stepsize,
	const float thresh
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	// 1 random number to select the level, 3 to select the position.
	rng.advance(i*4);
	uint32_t level = (uint32_t)(random_val(rng) * n_cascades) % n_cascades;

	// select grid cell that has density
	uint32_t idx;
	for (uint32_t j = 0; j < 10; ++j) {
		idx = ((i+step*n_elements) * 56924617 + j * 19349663 + 96925573) % n_grid_elements;
		idx += level * n_grid_elements;
		if (grid_in[idx] > thresh) {
			break;
		}
	}

	// random position within that cell
	uint32_t pos_idx = idx % n_grid_elements;

	uint32_t x = tcnn::morton3D_invert(pos_idx>>0);
	uint32_t y = tcnn::morton3D_invert(pos_idx>>1);
	uint32_t z = tcnn::morton3D_invert(pos_idx>>2);

	Vector3f pos = ((Vector3f{(float)x, (float)y, (float)z} + random_val_3d(rng)) / n_grid_size - Vector3f::Constant(0.5f)) * scalbnf(1.0f, level) + Vector3f::Constant(0.5f);

	out[i] = { warp_position(pos, aabb), warp_dt(min_cone_stepsize, min_cone_stepsize, n_cascades) };
	indices[i] = idx;
}

__global__ void splat_grid_samples_nerf_max_nearest_neighbor(
	const uint32_t n_elements, 
	const uint32_t* __restrict__ indices, 
	const int padded_output_width, 
	const tcnn::network_precision_t* network_output, 
	const EActivation density_activation,
	const uint32_t n_grid_size,
	const float min_cone_stepsize,
	float* __restrict__ grid_out
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	uint32_t local_idx = indices[i];

	// Current setting: optical thickness of the smallest possible stepsize.
	// Uncomment for:   optical thickness of the ~expected step size when the observer is in the middle of the scene
	uint32_t level = 0;//local_idx / (n_grid_size * n_grid_size * n_grid_size);

	float mlp = network_to_density(float(network_output[i * padded_output_width]), density_activation);
	float optical_thickness = mlp * scalbnf(min_cone_stepsize, level);

	// Positive floats are monotonically ordered when their bit pattern is interpretes as uint.
	// uint atomicMax is thus perfectly acceptable.
	atomicMax((uint32_t*)&grid_out[local_idx], __float_as_uint(optical_thickness));
}

__global__ void ema_grid_samples_nerf(
	const uint32_t n_elements,
	const float* __restrict__ grid_in,
	float decay,
	const uint32_t count,
	float* __restrict__ grid_out
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	float importance = grid_in[i];

	// float ema_debias_old = 1 - (float)powf(decay, count);
	// float ema_debias_new = 1 - (float)powf(decay, count+1);

	// float filtered_val = ((grid_out[i] * decay * ema_debias_old + importance * (1 - decay)) / ema_debias_new);
	// grid_out[i] = filtered_val;

	// Maximum instead of EMA allows capture of very thin features.
	// Basically, we want the grid cell turned on as soon as _ANYTHING_ visible is in there.

	float prev_val = grid_out[i];
	float val = (prev_val<0.f) ? prev_val : fmaxf(prev_val * decay, importance);
	grid_out[i] = val;
}

__global__ void grid_to_bitfield(
	const uint32_t n_elements,
	const float* __restrict__ grid,
	uint8_t* __restrict__ grid_bitfield,
	const float* __restrict__ mean_density_ptr,
	const float min_optical_thickness
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	uint8_t bits = 0;

	float thresh = std::min(min_optical_thickness, *mean_density_ptr);

	#pragma unroll
	for (uint8_t j = 0; j < 8; ++j) {
		bits |= grid[i*8+j] > thresh ? ((uint8_t)1 << j) : 0;
	}

	grid_bitfield[i] = bits;
}

__global__ void bitfield_max_pool(
	const uint32_t n_elements,
	const uint32_t n_grid_size,
	const uint8_t* __restrict__ prev_level,
	uint8_t* __restrict__ next_level
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	uint8_t bits = 0;

	#pragma unroll
	for (uint8_t j = 0; j < 8; ++j) {
		// If any bit is set in the previous level, set this
		// level's bit. (Max pooling.)
		bits |= prev_level[i*8+j] > 0 ? ((uint8_t)1 << j) : 0;
	}

	uint32_t x = tcnn::morton3D_invert(i>>0) + n_grid_size/8;
	uint32_t y = tcnn::morton3D_invert(i>>1) + n_grid_size/8;
	uint32_t z = tcnn::morton3D_invert(i>>2) + n_grid_size/8;

	next_level[tcnn::morton3D(x, y, z)] |= bits;
}

inline HOST_DEVICE float calc_dt(
	const float t, 
	const float cone_angle, 
	const float min_cone_stepsize,
	const float max_cone_stepsize
) {
	return tcnn::clamp(t*cone_angle, min_cone_stepsize, max_cone_stepsize);
}

inline __device__ int mip_from_pos(const Vector3f& pos, uint32_t n_cascades) {
	int exponent;
	float maxval = (pos - Vector3f::Constant(0.5f)).cwiseAbs().maxCoeff();
	frexpf(maxval, &exponent);
	return min(n_cascades-1, max(0, exponent+1));
}

inline __device__ int mip_from_dt(float dt, const Vector3f& pos, uint32_t n_grid_size, uint32_t n_cascades) {
	int mip = mip_from_pos(pos, n_cascades);
	dt *= 2 * n_grid_size;
	if (dt<1.f) return mip;
	int exponent;
	frexpf(dt, &exponent);
	return min(n_cascades-1, max(exponent, mip));
}

inline HOST_DEVICE uint32_t grid_mip_offset(uint32_t mip, uint32_t n_grid_size) {
	return (n_grid_size * n_grid_size * n_grid_size) * mip;
}

__device__ uint32_t cascaded_grid_idx_at(Vector3f pos, uint32_t mip, uint32_t n_grid_size) {
	float mip_scale = scalbnf(1.0f, -mip);
	pos -= Vector3f::Constant(0.5f);
	pos *= mip_scale;
	pos += Vector3f::Constant(0.5f);

	Vector3i i = (pos * n_grid_size).cast<int>();

	if (i.x() < -1 || i.x() > n_grid_size || i.y() < -1 || i.y() > n_grid_size || i.z() < -1 || i.z() > n_grid_size) {
		printf("WTF %d %d %d\n", i.x(), i.y(), i.z());
	}

	uint32_t idx = tcnn::morton3D(
		tcnn::clamp(i.x(), 0, (int)n_grid_size-1),
		tcnn::clamp(i.y(), 0, (int)n_grid_size-1),
		tcnn::clamp(i.z(), 0, (int)n_grid_size-1)
	);

	return idx;
}

__device__ bool density_grid_occupied_at(const Vector3f& pos, const uint8_t* density_grid_bitfield, uint32_t mip, uint32_t n_grid_size) {
	uint32_t idx = cascaded_grid_idx_at(pos, mip, n_grid_size);
	return density_grid_bitfield[idx/8+grid_mip_offset(mip,n_grid_size)/8] & (1<<(idx%8));
}

__device__ float cascaded_grid_at(Vector3f pos, const float* cascaded_grid, uint32_t mip, uint32_t n_grid_size) {
	uint32_t idx = cascaded_grid_idx_at(pos, mip, n_grid_size);
	return cascaded_grid[idx+grid_mip_offset(mip,n_grid_size)];
}

__device__ float& cascaded_grid_at(Vector3f pos, float* cascaded_grid, uint32_t mip, uint32_t n_grid_size) {
	uint32_t idx = cascaded_grid_idx_at(pos, mip, n_grid_size);
	return cascaded_grid[idx+grid_mip_offset(mip,n_grid_size)];
}

inline __device__ float distance_to_next_voxel(
	const Eigen::Vector3f& pos, 
	const Eigen::Vector3f& dir, 
	const Eigen::Vector3f& idir, 
	uint32_t res
) { // dda like step
	Eigen::Vector3f p = res * pos;
	float tx = (floorf(p.x() + 0.5f + 0.5f * copysignf(1.f, dir.x())) - p.x()) * idir.x();
	float ty = (floorf(p.y() + 0.5f + 0.5f * copysignf(1.f, dir.y())) - p.y()) * idir.y();
	float tz = (floorf(p.z() + 0.5f + 0.5f * copysignf(1.f, dir.z())) - p.z()) * idir.z();
	float t = min(min(tx, ty), tz);

	return fmaxf(t / res, 0.0f);
}

inline __device__ float advance_to_next_voxel(
	float t,
	const float cone_angle,
	const float min_cone_stepsize,
	const float max_cone_stepsize,
	const Eigen::Vector3f& pos,
	const Eigen::Vector3f& dir,
	const Eigen::Vector3f& idir,
	uint32_t res
) {
	// Analytic stepping by a multiple of dt. Make empty space unequal to non-empty space
	// due to the different stepping.
	// float dt = calc_dt(t, cone_angle, min_cone_stepsize, max_cone_stepsize);
	// return t + ceilf(fmaxf(distance_to_next_voxel(pos, dir, idir, res) / dt, 0.5f)) * dt;

	// Regular stepping (may be slower but matches non-empty space)
	float t_target = t + distance_to_next_voxel(pos, dir, idir, res);
	do {
		t += calc_dt(t, cone_angle, min_cone_stepsize, max_cone_stepsize);
	} while (t < t_target);
	return t;
}

__global__ void generate_training_samples_nerf(
	const uint32_t n_rays,
	const uint32_t n_rays_shift,
	const uint32_t n_coords,
	// dataset
	const uint32_t n_images,
	const Vector2i resolution,
	const Vector2f focal_length,
	const Vector2f principal_point,
	const Matrix<float, 3, 4>* __restrict__ training_xforms,
	const __half* __restrict__ training_images,
	// grid
	BoundingBox aabb,
	const uint32_t n_grid_size,
	const uint32_t n_cascades,
	const uint8_t* __restrict__ density_grid,
	// sampling parameters
	const bool snap_to_pixel_centers,
	const uint32_t n_max_steps,
	const float cone_angle_constant,
	const float min_cone_stepsize,
	const float max_cone_stepsize,
	const float near_distance,
	const float far_distance,
	const bool train_envmap,
	// output rays & coordinates
	uint32_t* __restrict__ ray_counter,				/* 1 */				// #sampled rays which having suitable sampled points
	uint32_t* __restrict__ numsteps_counter,		/* 1 */				// #sampled points of all rays
	Ray* __restrict__ rays_out,						/* n_rays */		// sampled rays
	Array4f* __restrict__ rays_rgba,				/* n_rays */		// ray rgba
	uint32_t* __restrict__ numsteps_out,			/* n_rays x 2 */ 	// first: #sampled points, second: start index of the sampled points
	Coordinate* __restrict__ coords_out,			/* n_coords */		// sampled points of all rays
	default_rng_t rng
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_rays) return;

	// sample the image index
	const uint32_t img = (((n_rays_shift + i) * n_images) / n_rays) % n_images;

	Matrix<float, 3, 4> xform = training_xforms[img];

	// sample the pixel location
	// 8 is used to index into the PRNG stream. Must be larger than the number of
	// samples consumed by any given training ray.
	rng.advance(i * 8);
	Vector2f xy = random_val_2d(rng);
	if (snap_to_pixel_centers) {
		xy = (xy.cwiseProduct(resolution.cast<float>()).cast<int>().cwiseMax(0).cwiseMin(resolution - Vector2i::Ones()).cast<float>() + Vector2f::Constant(0.5f)).cwiseQuotient(resolution.cast<float>());
	}

	// 1. generate the ray (i.e., the original and normalized view direction of the ray)
	Ray ray;
	// - rays need to be inferred from the camera matrix
	ray.o = xform.col(3);
	ray.d = {
		(xy.x()-principal_point.x())*resolution.x() / focal_length.x(),
		(xy.y()-principal_point.y())*resolution.y() / focal_length.y(),
		1.0f,
	};
	ray.d = (xform.block<3, 3>(0, 0) * ray.d).normalized();

	// 2. sample
	Vector2f tminmax = aabb.ray_intersect(ray.o, ray.d);
	// - the near distance prevents learning of camera-specific fudge right in front of the camera
	tminmax.x() = fmaxf(tminmax.x(), near_distance);

	// Pixel size. Doesn't always yield a good performance vs. quality
	// trade off. Especially if training pixels have a much different
	// size than rendering pixels.
	// return cosine*cosine / focal_length.mean();
	// calc_cone_angle(ray.d.dot(xform.col(2)), focal_length, cone_angle_constant);
	float cone_angle = cone_angle_constant;

	float startt = tminmax.x();
	startt += calc_dt(startt, cone_angle, min_cone_stepsize, max_cone_stepsize) * random_val(rng);
	Vector3f idir = ray.d.cwiseInverse();

	// - 2.1 first pass to compute an accurate number of steps
	uint32_t j = 0;
	float t = startt;
	Vector3f pos;
	// while(t < far_distance && j < n_max_steps) {
	// 	float dt = calc_dt(t, cone_angle, min_cone_stepsize, max_cone_stepsize);
	// 	++j;
	// 	t += dt;
	// }
	while (aabb.contains(pos = ray.o + t * ray.d) && j < n_max_steps) {
		float dt = calc_dt(t, cone_angle, min_cone_stepsize, max_cone_stepsize);
		uint32_t mip = mip_from_dt(dt, pos, n_grid_size, n_cascades);
		if (density_grid_occupied_at(pos, density_grid, mip, n_grid_size)) {
			++j;
			t += dt;
		} else {
			uint32_t res = n_grid_size>>mip;
			t = advance_to_next_voxel(t, cone_angle, min_cone_stepsize, max_cone_stepsize, pos, ray.d, idir, res);
		}
	}
	if (j == 0 && !train_envmap) {
		return;
	}
	uint32_t numsteps = j;
	// - first entry in the array is a counter
	uint32_t base = atomicAdd(numsteps_counter, numsteps);
	if (base + numsteps > n_coords) {
		return;
	}

	// - 2.1 second pass to generate sampled points
	coords_out += base;

	uint32_t ray_idx = atomicAdd(ray_counter, 1);

	rays_out[ray_idx] = ray;
	rays_rgba[ray_idx] = read_rgba(xy, resolution, img, training_images);
	numsteps_out[ray_idx*2+0] = numsteps;
	numsteps_out[ray_idx*2+1] = base;

	Vector3f warped_dir = warp_direction(ray.d);
	j = 0;
	t = startt;
	// while(t < far_distance && j < n_max_steps) {
	// 	float dt = calc_dt(t, cone_angle, min_cone_stepsize, max_cone_stepsize);
	// 	coords_out[j] = { warp_position(pos, aabb), warped_dir, warp_dt(dt, min_cone_stepsize, n_cascades) };
	// 	++j;
	// 	t += dt;
	// }
	while (aabb.contains(pos = ray.o + t * ray.d) && j < numsteps) {
		float dt = calc_dt(t, cone_angle, min_cone_stepsize, max_cone_stepsize);
		uint32_t mip = mip_from_dt(dt, pos, n_grid_size, n_cascades);
		if (density_grid_occupied_at(pos, density_grid, mip, n_grid_size)) {
			coords_out[j] = { warp_position(pos, aabb), warped_dir, warp_dt(dt, min_cone_stepsize, n_cascades) };
			++j;
			t += dt;
		} else {
			uint32_t res = n_grid_size>>mip;
			t = advance_to_next_voxel(t, cone_angle, min_cone_stepsize, max_cone_stepsize, pos, ray.d, idir, res);
		}
	}
}

__global__ void init_rays_with_payload_kernel_nerf(
	const Vector2i resolution,
	const Vector2f focal_length,
	const Vector2f principal_point,
	const Matrix<float, 3, 4> camera_matrix,
	const uint32_t spp,
	const bool snap_to_pixel_centers,
	const BoundingBox aabb,
	const network_precision_t* __restrict__ envmap_data,
	const Vector2i envmap_resolution,
	RayPayload* __restrict__ payloads,
	Array4f* __restrict__ framebuffer
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;
	if (x >= resolution.x() || y >= resolution.y()) return;

	// generate the ray of a pixel
	Ray ray = pixel_to_ray(
		{x, y},
		resolution,
		focal_length,
		principal_point,
		camera_matrix,
		spp,
		snap_to_pixel_centers
	);

	// determine whether the ray is alive
	const uint32_t idx = x + resolution.x() * y;
	RayPayload& payload = payloads[idx];

	// 0.05f is the nearest distance in rendering
	const float t = fmaxf(aabb.ray_intersect(ray.o, ray.d).x(), 0.05f) + 1e-6f;

	if (aabb.contains(ray.o + ray.d * t)) {
		payload.o = ray.o;
		payload.d = ray.d;
		payload.t = t;
		payload.idx = idx;
		payload.n_steps = 0;

		payload.alive = true;
	} else {
		payload.o = ray.o;

		payload.alive = false;
	}

	if (envmap_data) {
		framebuffer[idx] = read_envmap(envmap_data, envmap_resolution, ray.d);
	}
}

__global__ void advance_pos_nerf(
	const uint32_t n_rays,
	const uint32_t spp,
	RayPayload* __restrict__ payloads,
	BoundingBox render_aabb,
	const uint32_t n_grid_size,
	const uint32_t n_cascades,
	const uint8_t* __restrict__ density_grid,
	const float cone_angle_constant,
	const float min_cone_stepsize,
	const float max_cone_stepsize
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_rays) return;

	RayPayload& payload = payloads[i];
	if (!payload.alive) return;

	const Vector3f origin = payload.o;
	const Vector3f dir = payload.d;
	const Vector3f idir = dir.cwiseInverse();

	float t = payload.t;
	float dt = calc_dt(t, cone_angle_constant, min_cone_stepsize, max_cone_stepsize);
	t += ld_random_val(spp, i * 786433) * dt;
	Vector3f pos;

	while (true) {
		if (!render_aabb.contains(pos = origin + dir * t)) {
			payload.alive = false;
			break;
		}

		dt = calc_dt(t, cone_angle_constant, min_cone_stepsize, max_cone_stepsize);
		uint32_t mip = mip_from_dt(dt, pos, n_grid_size, n_cascades);

		if (!density_grid || density_grid_occupied_at(pos, density_grid, mip, n_grid_size)) {
			break;
		}

		uint32_t res = n_grid_size>>mip;
		t = advance_to_next_voxel(t, cone_angle_constant, min_cone_stepsize, max_cone_stepsize, pos, dir, idir, res);
	}

	payload.t = t;
}

__global__ void generate_coords_from_rays_regularly_kernel(
	const uint32_t n_rays,
	const uint32_t n_steps,
	const BoundingBox aabb,
	const uint32_t n_grid_size,
	const uint32_t n_cascades,
	const uint8_t* __restrict__ density_grid,
	const float cone_angle_constant,
	const float min_cone_stepsize,
	const float max_cone_stepsize,
	RayPayload* __restrict__ payloads,
	Coordinate* __restrict__ coords
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_rays) return;

	RayPayload& payload = payloads[i];
	if (!payload.alive) return;

	const Vector3f origin = payload.o;
	const Vector3f dir = payload.d;
	const Vector3f idir = dir.cwiseInverse();

	float t = payload.t;
	for (uint32_t j = 0; j < n_steps; ++j) {
		Vector3f pos;
		float dt = 0.0f;
		while (true) {
			// the coord is out of region-of-interest
			if (!aabb.contains(pos = origin + dir * t)) {
				payload.n_steps = j;
				return;
			}

			// occupancy grid guided
			dt = calc_dt(t, cone_angle_constant, min_cone_stepsize, max_cone_stepsize);
			uint32_t mip = mip_from_dt(dt, pos, n_grid_size, n_cascades);
			if (!density_grid || density_grid_occupied_at(pos, density_grid, mip, n_grid_size)) break;

			uint32_t res = n_grid_size>>mip;
			t = advance_to_next_voxel(t, cone_angle_constant, min_cone_stepsize, max_cone_stepsize, pos, dir, idir, res);
		}

		coords[i*n_steps+j] = { warp_position(pos, aabb), warp_direction(dir), warp_dt(dt, min_cone_stepsize, n_cascades) };
		t += dt;
	}
	payload.t = t;
	payload.n_steps = n_steps;
}


OccupancySampler::OccupancySampler(nlohmann::json& config) 
{
	this->aabb_scale		= config.value("aabb_scale", 1);
	this->aabb = BoundingBox{Vector3f::Constant(0.5f), Vector3f::Constant(0.5f)};
	this->aabb.inflate(0.5f * this->aabb_scale);

	this->n_grid_size 		= config.value("grid_size", 128);
	this->n_grid_elements 	= this->n_grid_size * this->n_grid_size * this->n_grid_size;
	this->n_cascades		= 2;
	while (1 << (this->n_cascades-1) < this->aabb_scale) this->n_cascades++;
	this->n_total_elements	= this->n_grid_elements * this->n_cascades;

	this->ema_decay 		= config.value("ema_decay", 0.95f);
	
	this->density_grid_current.resize(this->n_total_elements);
	this->density_grid.resize(this->n_total_elements);
	this->density_grid_mean.resize(tcnn::reduce_sum_workspace_size(this->n_grid_elements));
	this->density_grid_bitfield.resize(this->n_total_elements / 8);
	
	this->n_max_steps 			= config.value("maximum_marching_steps", 1024);
	this->cone_angle_constant	= config.value("cone_angle_constant", 0.f);
	this->min_cone_stepsize		= SQRT3() / this->n_max_steps;
	this->max_cone_stepsize 	= SQRT3() * (1<<(this->n_cascades-1)) / this->n_grid_size;
	this->near_distance			= config.value("near_distance", 0.2f);
	this->far_distance			= config.value("far_distance", 100.f);
	this->snap_to_pixel_centers_in_training	= config.value("snap_to_pixel_centers_in_training", true);
	this->snap_to_pixel_centers_in_rendering = config.value("snap_to_pixel_centers_in_rendering", true);
	this->min_optical_thickness = config.value("min_optical_thickness", 0.01f);
}

void OccupancySampler::init(
	const uint32_t n_images,
	const Vector2i& image_resolution,
	const Vector2f& focal_length,
	const Vector2f& principal_point,
	const GPUMemory<Matrix<float, 3, 4>>& camera_matrices,
	const GPUMemory<__half>& images,
	default_rng_t& rng,
	hipStream_t stream
) {
	this->i_step = 0;
    this->rng = default_rng_t{rng.next_uint()};

	// only cull away empty regions where no camera is looking when the cameras are actually meaningful.
	linear_kernel(mark_untrained_density_grid, 0, stream,
		this->n_total_elements,
		this->n_grid_size,
		n_images,
		image_resolution,
		focal_length,
		camera_matrices.data(),
		this->density_grid.data()
	);

	// CUDA_CHECK_THROW(hipMemsetAsync(this->density_grid.data(), 0, this->density_grid.get_bytes(), stream));
}

void OccupancySampler::sample_positions_from_grid(
	const uint32_t n_uniform_positions,
	const uint32_t n_nonuniform_positions,
	GPUMemory<Position>& grid_positions,
	GPUMemory<uint32_t>& grid_indices,
	hipStream_t stream
) {
	const uint32_t n_total_positions = n_uniform_positions + n_nonuniform_positions;

	grid_positions.enlarge(n_total_positions);
	grid_indices.enlarge(n_total_positions);

	linear_kernel(generate_grid_samples_nerf_nonuniform, 0, stream,
		n_uniform_positions,
		this->rng,
		this->i_step,
		this->aabb,
		this->density_grid.data(),
		grid_positions.data(),
		grid_indices.data(),
		this->n_grid_size,
		this->n_grid_elements,
		this->n_cascades,
		this->min_cone_stepsize,
		-0.01
	);
	this->rng.advance();

	linear_kernel(generate_grid_samples_nerf_nonuniform, 0, stream,
		n_nonuniform_positions,
		this->rng,
		this->i_step,
		this->aabb,
		this->density_grid.data(),
		grid_positions.data() + n_uniform_positions,
		grid_indices.data() + n_uniform_positions,
		this->n_grid_size,
		this->n_grid_elements,
		this->n_cascades,
		this->min_cone_stepsize,
		this->min_optical_thickness
	);
	this->rng.advance();
}

void OccupancySampler::update_grid(
	const uint32_t n_total_positions,
	const uint32_t padded_output_width,
	GPUMemory<precision_t>& grid_densities,
	GPUMemory<uint32_t>& grid_indices,
	EActivation density_activation,
	hipStream_t stream
) {
	CUDA_CHECK_THROW(hipMemsetAsync(this->density_grid_current.data(), 0, this->density_grid_current.get_bytes(), stream));

	linear_kernel(splat_grid_samples_nerf_max_nearest_neighbor, 0, stream,
		n_total_positions,
		grid_indices.data(),
		padded_output_width,
		grid_densities.data(),
		density_activation,
		this->n_grid_size,
		this->min_cone_stepsize,
		this->density_grid_current.data()
	);
	
	linear_kernel(ema_grid_samples_nerf, 0, stream,
		this->n_total_elements,
		this->density_grid_current.data(),
		this->ema_decay,
		this->i_step,
		this->density_grid.data()
	);

	++this->i_step;
}

void OccupancySampler::update_grid_bitfield(hipStream_t stream)
{
	CUDA_CHECK_THROW(hipMemsetAsync(this->density_grid_mean.data(), 0, sizeof(float), stream));
	const uint32_t n_elements = this->n_grid_elements;
	reduce_sum(
		this->density_grid.data(), 
		[n_elements] __device__ (float val) { return fmaxf(val, 0.f) / (n_elements); }, 
		this->density_grid_mean.data(), this->n_grid_elements, stream);

	linear_kernel(grid_to_bitfield, 0, stream, 
		this->n_total_elements / 8,
		this->density_grid.data(), 
		this->density_grid_bitfield.data(),
		this->density_grid_mean.data(),
		this->min_optical_thickness);

	for (uint32_t level = 1; level < this->n_cascades; ++level) {
		linear_kernel(bitfield_max_pool, 0, stream, 
			this->n_grid_elements / 64,
			this->n_grid_size,
			this->density_grid_bitfield.data() + this->n_grid_elements * (level-1) / 8,
			this->density_grid_bitfield.data() + this->n_grid_elements * level / 8);
	}
}

void OccupancySampler::sample_coords_from_dataset(
	const uint32_t n_rays,
	const uint32_t n_rays_shift,
	const uint32_t n_coords,
	const NeRFSynthetic& dataset,
	const bool train_envmap,
	tcnn::GPUMemory<Ray>& rays, 
	tcnn::GPUMemory<Eigen::Array4f>& rays_rgba,
	uint32_t* __restrict__ n_rays_counter,
	tcnn::GPUMemory<uint32_t>& ray_coord_indices,
	tcnn::GPUMemory<Coordinate>& coords,
	uint32_t* __restrict__ n_coords_counter,
	tcnn::default_rng_t rng,
	hipStream_t stream
) {
	linear_kernel(generate_training_samples_nerf, 0, stream,
		n_rays,
		n_rays_shift,
		n_coords,
		// dataset
		dataset.n_images,
		dataset.image_resolution,
		dataset.focal_length,
		dataset.principal_point,
		dataset.xforms.data(),
		dataset.images.data(),
		// grid
		this->aabb,
		this->n_grid_size,
		this->n_cascades,
		this->density_grid_bitfield.data(),
		// sampling parameters
		this->snap_to_pixel_centers_in_training,
		this->n_max_steps,
		this->cone_angle_constant,
		this->min_cone_stepsize,
		this->max_cone_stepsize,
		this->near_distance,
		this->far_distance,
		train_envmap,
		// output rays & coordinates
		n_rays_counter,
		n_coords_counter,
		rays.data(),
		rays_rgba.data(),
		ray_coord_indices.data(),
		coords.data(),
		rng
	);
}

void OccupancySampler::generate_rays_from_camera_matrix(
	const Vector2i& resolution, 
	const Vector2f& focal_length, 
	const Vector2f& principal_point, 
	const Matrix<float, 3, 4>& camera_matrix, 
	const precision_t* __restrict__ envmap_data,
	const Vector2i& envmap_resolution,
	GPUMemory<RayPayload>& payload,
	CudaRenderBuffer& render_buffer, 
	hipStream_t stream
) {
	const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { 
		div_round_up((uint32_t)resolution.x(), threads.x), 
		div_round_up((uint32_t)resolution.y(), threads.y), 
		1 };
	init_rays_with_payload_kernel_nerf<<<blocks, threads, 0, stream>>>(
		resolution,
		focal_length,
		principal_point,
		camera_matrix,
		render_buffer.spp(),
		this->snap_to_pixel_centers_in_rendering,
		this->aabb,
		envmap_data,
		envmap_resolution,
		payload.data(),
		render_buffer.frame_buffer()
	);

	linear_kernel(advance_pos_nerf, 0, stream,
		resolution.x() * resolution.y(),
		render_buffer.spp(),
		payload.data(),
		this->aabb,
		this->n_grid_size,
		this->n_cascades,
		this->density_grid_bitfield.data(),
		this->cone_angle_constant,
		this->min_cone_stepsize,
		this->max_cone_stepsize
	);
}

void OccupancySampler::generate_coords_from_rays(
	const uint32_t n_rays,
	const uint32_t n_steps,
	tcnn::GPUMemory<RayPayload> payloads,
	tcnn::GPUMemory<Coordinate> coords,
	hipStream_t stream
) {
	linear_kernel(generate_coords_from_rays_regularly_kernel, 0, stream,
		n_rays,
		n_steps,
		this->aabb,
		this->n_grid_size,
		this->n_cascades,
		this->density_grid_bitfield.data(),
		this->cone_angle_constant,
		this->min_cone_stepsize,
		this->max_cone_stepsize,
		payloads.data(),
		coords.data()
	);
}