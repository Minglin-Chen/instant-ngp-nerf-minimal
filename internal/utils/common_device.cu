#include "hip/hip_runtime.h"
/** @author	Minglin Chen
 *  @date 	2023/3/23
 *  @Email 	chenmlin8@mail2.sysu.edu.cn
 */

#include "internal/utils/common.h"
#include "internal/utils/common_device.h"

#include <unsupported/Eigen/MatrixFunctions>
#include <stb_image/stb_image.h>

using namespace Eigen;
using namespace tcnn;


Matrix<float, 3, 4> spec_opengl_to_opencv(const Matrix<float, 3, 4> ogl_matrix, const float scale, const Vector3f offset) {
	Matrix<float, 3, 4> ocv_matrix = ogl_matrix;
	
	ocv_matrix.col(1) *= -1.0f;
	ocv_matrix.col(2) *= -1.0f;
	ocv_matrix.col(3) = ocv_matrix.col(3) * scale + offset;

	// cycle axes: xyz <- yzx
	Vector4f tmp = ocv_matrix.row(0);
	ocv_matrix.row(0) = (Vector4f)ocv_matrix.row(1);
	ocv_matrix.row(1) = (Vector4f)ocv_matrix.row(2);
	ocv_matrix.row(2) = tmp;

	return ocv_matrix;
}

Matrix<float, 3, 4> log_space_lerp(const Matrix<float, 3, 4>& begin, const Matrix<float, 3, 4>& end, float t) {
	Matrix4f A = Matrix4f::Identity();
	A.block<3,4>(0,0) = begin;
	Matrix4f B = Matrix4f::Identity();
	B.block<3,4>(0,0) = end;

	Matrix4f log_space_a_to_b = (B * A.inverse()).log();

	return ((log_space_a_to_b * t).exp() * A).block<3,4>(0,0);
}

GPUMemory<float> load_stbi(const std::string& filename, int& width, int& height) {
	bool is_hdr = stbi_is_hdr(filename.c_str());

	void* data; // width * height * RGBA
	int comp;
	if (is_hdr) {
		data = stbi_loadf(filename.c_str(), &width, &height, &comp, 4);
	} else {
		data = stbi_load(filename.c_str(), &width, &height, &comp, 4);
	}

	if (!data) {
		throw std::runtime_error{std::string{stbi_failure_reason()}};
	}

	ScopeGuard mem_guard{[&]() { stbi_image_free(data); }};

	if (width == 0 || height == 0) {
		throw std::runtime_error{"Image has zero pixels."};
	}

	GPUMemory<float> result(width * height * 4);
	if (is_hdr) {
		result.copy_from_host((float*)data);
	} else {
		GPUMemory<uint8_t> bytes(width * height * 4);
		bytes.copy_from_host((uint8_t*)data);
		linear_kernel(from_rgba32<float>, 0, nullptr, width * height, bytes.data(), result.data());
	}

	return result;
}